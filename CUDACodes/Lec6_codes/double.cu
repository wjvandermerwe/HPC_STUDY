#include "hip/hip_runtime.h"
/*
  This example is using hipMalloc and hipMemcpy
  to transfer data between host and device explicitly.
  To compile: nvcc -I../common double.cu -o double
  To run: ./double

  upon run, the following code outputs:
    All elements were doubled? FALSE
  find the error(s) in the code and get output as
    All elements were doubled? TRUE
*/
#include <stdio.h>
#include <hip/hip_runtime_api.h>

void init(int *a, int N) {
  int i;
  for (i = 0; i < N; ++i) {
    a[i] = i;
  }
}

__global__ void doubleElements(int *a, int N) {
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i <= N) {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N) {
  int i;
  for (i = 0; i < N; ++i) {
    if (a[i] != i*2)
    {
       printf("a[%d] = %d, should be %d\n", i, a[i], i*2);
       return false;
    }
  }
  return true;
}

int main() {
  int N = 10000;
  //int *a;
  int *a, *host_a;

  size_t size = N * sizeof(int);

  //checkCudaErrors(hipMallocManaged(&a, size));
  //init(a, N);

  host_a = (int *)malloc(size);
  checkCudaErrors(hipMalloc((void**)&a, size));
  init(host_a, N);
  checkCudaErrors(hipMemcpy(a, host_a, size, hipMemcpyHostToDevice));


  //the size of this grid is 256*32 = 8192.
  size_t threads_per_block = 250;
  size_t number_of_blocks = 40;
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  getLastCudaError("doubleElements execution failed\n");
  hipDeviceSynchronize();
  checkCudaErrors(hipMemcpy(host_a, a, size, hipMemcpyDeviceToHost));
  bool areDoubled = checkElementsAreDoubled(host_a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");
  free(host_a);
  checkCudaErrors(hipFree(a));
  hipDeviceReset();
  return 0;
}
